#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "Solution1.cu"
// #include "Solution2.cu"
// #include "Solution3.cu"

/**
* Error checking function;
*/
#define gpuErrchk(ans) { gpuAssert((ans), __LINE__); }
inline void gpuAssert(hipError_t code, int line, bool abort = false, bool log = true) {
    if (log && code != hipSuccess) {
        fprintf(stderr, "GPUassert: line %d - %s\n", line, hipGetErrorString(code));
        if (abort)
            exit(code);
    }
}

bool your_solution(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {


    /**
    * @input ref reference points
    * @input ref_nb number of reference points
    * @input query the data to process and classify
    * @input query_nb number of query
    * @input dim dimension of every single point
    * @input k number of neighbors
    * @input knn_dist array to save the distances between every query 
             and the reference point
    * @input knn_index array with the solution of the classification
    */

    std::cout << "\nStarting Gpu function\n";

    // ---------------------------------- Variables' declaration ------------------------------- 

    int block_size = 1024;
    int grid = (query_nb + block_size -1)/block_size;    

    // ---------------------------------- Creating data location on gpu -------------------------------
    // Location for all reference data
    float * ref_gpu;
    gpuErrchk(hipMallocManaged(&ref_gpu, ref_nb*dim*sizeof(float)));

    // Location for all query data
    float * query_gpu;
    gpuErrchk(hipMallocManaged(&query_gpu, query_nb*dim*sizeof(float)));

    // Location for the k-nearest distances
    float * knn_dist_gpu;
    gpuErrchk(hipMallocManaged(&knn_dist_gpu, query_nb*k*sizeof(float)));

    // Location for the k-nearest index
    int * knn_index_gpu;
    gpuErrchk(hipMallocManaged(&knn_index_gpu, query_nb*k*sizeof(int)));

    // Location for index and dist 
    int * index_gpu;
    gpuErrchk(hipMallocManaged(&index_gpu, query_nb*ref_nb*sizeof(int)));
    float * dist_gpu;
    gpuErrchk(hipMallocManaged(&dist_gpu, query_nb*ref_nb*sizeof(float)));

    // ---------------------------------- Transfering data on device -------------------------------

    gpuErrchk(hipMemcpy(ref_gpu, ref, ref_nb*dim*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(query_gpu, query, query_nb*dim*sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipDeviceSynchronize());

    // ---------------------------------- Kernel launching -------------------------------

    // Solution - 1
    //knn_gpu_1_Block_Grid<<<1, 1>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu);
    
    // Solution - 2
    //knn_gpu<<<grid, block_size>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu);
    
    // Solution - 3
    knn_gpu_v2<<<grid, block_size>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu, index_gpu, dist_gpu);
    
    // ---------------------------------- Transfering data on host -------------------------------

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(knn_dist, knn_dist_gpu, query_nb*k*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(knn_index, knn_index_gpu, query_nb*k*sizeof(int), hipMemcpyDeviceToHost));

    // ---------------------------------- Debug section -------------------------------

    
    for(int i = 0; i < query_nb; ++i){
        std::cout<< i <<" query:" << std::endl; 

        for (int j = 0; j < k; ++j){
            std::cout << "\treference index: " << knn_index[i + j] << " dist: " << knn_dist[i + j] <<std::endl;
        }
    }
    
    

    // ---------------------------------- Free memory -------------------------------

    hipFree(ref_gpu);
    hipFree(query_gpu);
    hipFree(knn_dist_gpu);
    hipFree(knn_index_gpu);

    hipFree(index_gpu);
    hipFree(dist_gpu);
    
    return true;
}