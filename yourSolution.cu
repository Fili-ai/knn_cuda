#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "variables.h"

//#include "Solution1.cu"
#include "Solution2.cu"
//#include "Solution4.cu"

/**
* Error checking function;
*/
#define gpuErrchk(ans) { gpuAssert((ans), __LINE__); }
inline void gpuAssert(hipError_t code, int line, bool abort = false) {
    if (LOG_LEVEL < 3 && code != hipSuccess) {
        fprintf(stderr, "GPUassert: line %d - %s\n", line, hipGetErrorString(code));
        if (abort)
            exit(code);
    }
}

bool your_solution(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {


    /**
    * @input ref reference points
    * @input ref_nb number of reference points
    * @input query the data to process and classify
    * @input query_nb number of query
    * @input dim dimension of every single point
    * @input k number of neighbors
    * @input knn_dist array to save the distances between every query 
             and the reference point
    * @input knn_index array with the solution of the classification
    */

    // ---------------------------------- Variables' declaration ------------------------------- 

    dim3 block_size(1024, 1, 1);
    dim3 grid((query_nb + block_size.x - 1) / block_size.x, 1, 1);
    
    dim3 block_size_cosine_distance(1024, 1, 1);
    dim3 grid_cosine_distance((ref_nb*query_nb + block_size.x - 1) / block_size.x, 1, 1);

    // ---------------------------------- Creating data location on gpu -------------------------------
    // Location for all reference data
    float * ref_gpu;
    gpuErrchk(hipMalloc(&ref_gpu, ref_nb*dim*sizeof(float)));

    // Location for all query data
    float * query_gpu;
    gpuErrchk(hipMalloc(&query_gpu, query_nb*dim*sizeof(float)));

    // Location for the k-nearest distances
    float * knn_dist_gpu;
    gpuErrchk(hipMalloc(&knn_dist_gpu, query_nb*k*sizeof(float)));

    // Location for the k-nearest index
    int * knn_index_gpu;
    gpuErrchk(hipMalloc(&knn_index_gpu, query_nb*k*sizeof(int)));

    // Location for index and dist 
    int * index_gpu;
    gpuErrchk(hipMalloc(&index_gpu, query_nb*ref_nb*sizeof(int)));
    float * dist_gpu;
    gpuErrchk(hipMalloc(&dist_gpu, query_nb*ref_nb*sizeof(float)));
    

    // ---------------------------------- Transfering data on device -------------------------------

    gpuErrchk(hipMemcpy(ref_gpu, ref, ref_nb*dim*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(query_gpu, query, query_nb*dim*sizeof(float), hipMemcpyHostToDevice));

    gpuErrchk(hipDeviceSynchronize());

    // ---------------------------------- Kernel launching -------------------------------

    // Solution - 1
    //knn_gpu_1_Block_Grid<<<1, 1>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu);
     
    // Solution - 2
    knn_gpu<<<grid, block_size>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu, index_gpu, dist_gpu);

    
    // Solution - 4
    // cosine_distance_gpu<<<grid_cosine_distance, block_size_cosine_distance>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, dist_gpu, index_gpu);
    // insertion_sort_gpu<<<grid, block_size>>>(ref_nb, query_nb, dim, k, knn_dist_gpu, knn_index_gpu, index_gpu, dist_gpu);


    // ---------------------------------- Transfering data on host -------------------------------

    gpuErrchk(hipDeviceSynchronize());

    gpuErrchk(hipMemcpy(knn_dist, knn_dist_gpu, query_nb*k*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(knn_index, knn_index_gpu, query_nb*k*sizeof(int), hipMemcpyDeviceToHost));

    // ---------------------------------- Debug section -------------------------------

    
    if(LOG_LEVEL < 2){   
        for(int i = 0; i < query_nb ; ++i){
            std::cout<< std::endl << i <<" query:" << std::endl; 
            for (int j = 0; j < ref_nb; ++j){
                std::cout << "\treference index: " << index_gpu[i + j*query_nb] << " dist: " << dist_gpu[i + j*query_nb] <<std::endl;
            }
        }
    }

    
    // ---------------------------------- Free memory -------------------------------

    hipFree(ref_gpu);
    hipFree(query_gpu);
    hipFree(knn_dist_gpu);
    hipFree(knn_index_gpu);

    hipFree(index_gpu);
    hipFree(dist_gpu);
    
    return true;
}