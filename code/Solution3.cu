#pragma once
#include <hip/hip_runtime.h>


__global__ void cosine_distance_gpu(const float * ref,
                                    const int     ref_nb,
                                    const float * query,
                                    const int     query_nb,
                                    const int     dim,
                                    float *       dist,
                                    int *         index,  
                                    const int     chunk) {
    
    /**
     * @brief function to calculate the cosine distance of references and queries
     * @param ref array containing all references
     * @param ref_nb number of references
     * @param query array containing all queries
     * @param query_nb number of queries
     * @param dim dimension of each point (same for queries and references)
     * @param index array containing all reference's indexes
     * @param dist array containing all reference's distances
     * @param chunk number of query processed per iteration
    */

    // unique id of a thread
    int unique_id = blockIdx.x * blockDim.x + threadIdx.x;
    
    // query and reference index on which the thread have to work 
    int query_index = unique_id / (ref_nb);
    int ref_index = unique_id % ref_nb;
    
    //cosine distance calculations
    if(query_index < query_nb){

        double dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;

        #pragma unroll
        for(unsigned int d = 0u; d < dim; ++d) {
            dot += ref[d * ref_nb + ref_index] * query[d * chunk + query_index];
            denom_a += ref[d * ref_nb + ref_index] * ref[d * ref_nb + ref_index] ;
            denom_b += query[d * chunk + query_index] * query[d * chunk + query_index] ;
        } 

        index[query_index + ref_index*chunk] = ref_index;
        dist[query_index + ref_index*chunk] = dot / (sqrt(denom_a) * sqrt(denom_b));

    }  
}

__global__ void insertion_sort_gpu( const int     ref_nb,
                                    const int     query_nb,
                                    const int     dim,
                                    const int     k,
                                    float *       knn_dist,
                                    int *         knn_index,
                                    const int *   index, 
                                    const float * dist){

    /**
     * @brief Insertion sort for the reference's distances
     * @param ref_nb number of references
     * @param query_nb number of queries
     * @param dim dimension of each element (same for each query and reference)
     * @param k number of items of interest
     * @param knn_index array to store the first k reference's indexes
     * @param knn_dist array to store the first k reference's distances
     * @param index array with the index of the reference distance
     * @param dist array with distances to sort
    */

    // unique id of a thread
    int query_index = blockIdx.x * blockDim.x + threadIdx.x;

    if(query_index < query_nb){
        // Allocate local array to store all the distances / indexes for a given query point 
        float * dist_sorted  = (float *) malloc((k+1) * sizeof(float));
        int *   index_sorted = (int *)   malloc((k+1) * sizeof(int));
        float curr_dist;
        int  curr_index;
        
        for (int i=0; i<ref_nb; ++i) {

            // Store current distance and associated index
            curr_dist  = dist[query_index + i * query_nb];
            curr_index = index[query_index + i * query_nb];     
            
            // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
            if (i >= k && curr_dist >= dist_sorted[k-1]) {
                continue;
            }

            // Shift values (and indexes) higher that the current distance to the right
            int j = i < k-1 ? i : k-1;  
            while (j >= 0 && dist_sorted[j-1] > curr_dist) {
                dist_sorted[j]  = dist_sorted[j-1];
                index_sorted[j] = index_sorted[j-1];
                --j;
            }
        
            // Write the current distance and index at their position
            dist_sorted[j]  = curr_dist;
            index_sorted[j] = curr_index; 
                
        }
        

        for(int i = 0; i < k; ++i){
            // to save the k distances at distance query_nb
            knn_index[query_index + i * query_nb] = index_sorted[i];
            knn_dist[query_index + i * query_nb] = dist_sorted[i];
        }

        free(dist_sorted);
        free(index_sorted); 
    }
}