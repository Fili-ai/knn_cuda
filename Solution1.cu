#pragma once
#include <hip/hip_runtime.h>


/**
 * Insertion sort to sort ref distances
*/
__device__ void insertion_sort_gpu(float *dist_sort, int *index_sort, int length, int k){
    // Initialise the first index
    index_sort[0] = 0;

    // Go through all points
    for (int i=1; i<length; ++i) {

        // Store current distance and associated index
        float curr_dist  = dist_sort[i];
        int   curr_index = i;

        // Skip the current value if its index is >= k and if it's higher the k-th slready sorted mallest value
        if (i >= k && curr_dist >= dist_sort[k-1]) {
            continue;
        }

        // Shift values (and indexes) higher that the current distance to the right
        //int j = std::min(i, k-1);
        int j = i < k-1 ? i : k-1; 
        while (j > 0 && dist_sort[j-1] > curr_dist) {
            dist_sort[j]  = dist_sort[j-1];
            index_sort[j] = index_sort[j-1];
            --j;
        }

        // Write the current distance and index at their position
        dist_sort[j]  = curr_dist;
        index_sort[j] = curr_index; 
    }
}

/**
 * Cosine distance between ref and query
 */
__device__ float cosine_distance_gpu(const float * ref,
                       int           ref_nb,
                       const float * query,
                       int           query_nb,
                       int           dim,
                       int           ref_index,
                       int           query_index) {

    double dot = 0.0, denom_a = 0.0, denom_b = 0.0 ;
    for(unsigned int d = 0u; d < dim; ++d) {
        dot += ref[d * ref_nb + ref_index] * query[d * query_nb + query_index] ;
        denom_a += ref[d * ref_nb + ref_index] * ref[d * ref_nb + ref_index] ;
        denom_b += query[d * query_nb + query_index] * query[d * query_nb + query_index] ;
    }
    return dot / (sqrt(denom_a) * sqrt(denom_b)) ;
}

/**
 * Kernel to solve our problem. It elaborate all queries
*/
__global__ void knn_gpu(const float *  ref,
                        int           ref_nb,
                        const float * query,
                        int           query_nb,
                        int           dim,
                        int           k,
                        float *       knn_dist,
                        int *         knn_index){

    // Allocate local array to store all the distances / indexes for a given query point 
    float * dist  = (float *) malloc(ref_nb * sizeof(float));
    int *   index = (int *)   malloc(ref_nb * sizeof(int));

    // Process one query point at the time
    for (int i=0; i<query_nb; ++i) {

        // Compute all distances / indexes
        for (int j=0; j<ref_nb; ++j) {
            dist[j]  = cosine_distance_gpu(ref, ref_nb, query, query_nb, dim, j, i);
            index[j] = j;
        }

        // Sort distances / indexes
        insertion_sort_gpu(dist, index, ref_nb, k);

        // Copy k smallest distances and their associated index
        for (int j=0; j<k; ++j) {
            knn_dist[j * query_nb + i]  = dist[j];
            knn_index[j * query_nb + i] = index[j];
        }
    }

    free(index);
    free(dist);
}