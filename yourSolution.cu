#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "variables.h"

//#include "Solution1.cu"
//#include "Solution2.cu"
//#include "Solution4.cu"
#include "Solution5.cu"

/**
* Error checking function;
*/
#define gpuErrchk(ans) { gpuAssert((ans), __LINE__); }
inline void gpuAssert(hipError_t code, int line, bool abort = false) {
    if (LOG_LEVEL < 3 && code != hipSuccess) {
        fprintf(stderr, "GPUassert: line %d - %s\n", line, hipGetErrorString(code));
        if (abort)
            exit(code);
    }
}

bool your_solution(const float * ref,
                     int           ref_nb,
                     const float * query,
                     int           query_nb,
                     int           dim,
                     int           k,
                     float *       knn_dist,
                     int *         knn_index) {


    /**
    * @input ref reference points
    * @input ref_nb number of reference points
    * @input query the data to process and classify
    * @input query_nb number of query
    * @input dim dimension of every single point
    * @input k number of neighbors
    * @input knn_dist array to save the distances between every query 
             and the reference point
    * @input knn_index array with the solution of the classification
    */

    // ---------------------------------- Creating data location on gpu -------------------------------
    // Location for all reference data
    float * ref_gpu;
    gpuErrchk(hipMalloc(&ref_gpu, ref_nb*dim*sizeof(float)));

    // Location for all query data
    float * query_gpu;
    gpuErrchk(hipMalloc(&query_gpu, query_nb*dim*sizeof(float)));

    // Location for the k-nearest distances
    float * knn_dist_gpu;
    gpuErrchk(hipMalloc(&knn_dist_gpu, query_nb*k*sizeof(float)));

    // Location for the k-nearest index
    int * knn_index_gpu;
    gpuErrchk(hipMalloc(&knn_index_gpu, query_nb*k*sizeof(int)));

    // Location for index and dist 
    int * index_gpu;
    gpuErrchk(hipMalloc(&index_gpu, query_nb*ref_nb*sizeof(int)));
    float * dist_gpu;
    gpuErrchk(hipMalloc(&dist_gpu, query_nb*ref_nb*sizeof(float)));
    

    // ---------------------------------- Transfering data on device -------------------------------

    gpuErrchk(hipMemcpy(ref_gpu, ref, ref_nb*dim*sizeof(float), hipMemcpyHostToDevice));
    gpuErrchk(hipMemcpy(query_gpu, query, query_nb*dim*sizeof(float), hipMemcpyHostToDevice));

    //gpuErrchk(hipDeviceSynchronize());

    // ---------------------------------- Kernel launching -------------------------------

    // Solution - 1
    //knn_gpu_1_Block_Grid<<<1, 1>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu);
     
    // Solution - 2
    //dim3 block_size(1024, 1, 1);
    //dim3 grid((query_nb + block_size.x - 1) / block_size.x, 1, 1);
    //
    //knn_gpu<<<grid, block_size>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, k, knn_dist_gpu, knn_index_gpu, index_gpu, dist_gpu);
 
    // Solution - 4
    //dim3 block_size(256, 1, 1);
    //dim3 grid((query_nb + block_size.x - 1) / block_size.x, 1, 1);
    //dim3 block_size_cosine_distance(256, 1, 1);
    //dim3 grid_cosine_distance((ref_nb*query_nb + block_size.x - 1) / block_size.x, 1, 1);
    //cosine_distance_gpu<<<grid_cosine_distance, block_size_cosine_distance>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, dist_gpu, index_gpu);
    //insertion_sort_gpu<<<grid, block_size>>>(ref_nb, query_nb, dim, k, knn_dist_gpu, knn_index_gpu, index_gpu, dist_gpu);
    
    // Solution - 5
    // memory management 
    float * dots;
    gpuErrchk(hipMallocManaged(&dots, query_nb*ref_nb*dim*sizeof(float)));
    float * denom_a;
    gpuErrchk(hipMallocManaged(&denom_a, query_nb*ref_nb*dim*sizeof(float)));
    float * denom_b;
    gpuErrchk(hipMallocManaged(&denom_b, query_nb*ref_nb*dim*sizeof(float)));
    float * sum_dots;
    gpuErrchk(hipMallocManaged(&sum_dots, query_nb*ref_nb*sizeof(float)));
    float * sum_denom_a;
    gpuErrchk(hipMallocManaged(&sum_denom_a, query_nb*ref_nb*sizeof(float)));
    float * sum_denom_b;
    gpuErrchk(hipMallocManaged(&sum_denom_b, query_nb*ref_nb*sizeof(float)));

    //// block and grid dimension 
    dim3 block_size(1024, 1, 1);
    dim3 grid((query_nb + block_size.x - 1) / block_size.x, 1, 1);
    dim3 block_size_fill(1024, 1, 1);
    dim3 grid_fill((ref_nb*query_nb*dim + block_size.x - 1) / block_size.x, 1, 1);
    dim3 block_size_reduction(1024, 1, 1);
    dim3 grid_reduction((ref_nb*query_nb + block_size.x - 1) / block_size.x, 1, 1); 
    dim3 block_size_cosine_distance(1024, 1, 1);
    dim3 grid_cosine_distance((ref_nb*query_nb + block_size.x - 1) / block_size.x, 1, 1);

    //// kernel launching
    fill_gpu<<<grid_fill, block_size_fill>>>(ref_gpu, ref_nb, query_gpu, query_nb, dim, dots, denom_a, denom_b);
    //reduceDimension<<<grid_reduction, block_size_reduction>>>(dots, denom_a, denom_b, ref_nb, query_nb, dim, sum_dots, sum_denom_a, sum_denom_b); 
    reduce0<<<grid_fill, block_size_fill, block_size.x * sizeof(float)>>>(dots, sum_dots, query_nb, ref_nb, dim);
    reduce0<<<grid_fill, block_size_fill, block_size.x * sizeof(float)>>>(denom_a, sum_denom_a, query_nb, ref_nb, dim);
    reduce0<<<grid_fill, block_size_fill, block_size.x * sizeof(float)>>>(denom_b, sum_denom_b, query_nb, ref_nb, dim);
    hipFree(dots);
    hipFree(denom_a);
    hipFree(denom_b);
    cosine_distance_gpu<<<grid_cosine_distance, block_size_cosine_distance>>>(ref_nb, query_nb, dist_gpu, index_gpu, sum_dots, sum_denom_a, sum_denom_b);
    //hipFree(sum_dots);
    //hipFree(sum_denom_a);
    //hipFree(sum_denom_b);
    insertion_sort_gpu<<<grid, block_size>>>(ref_nb, query_nb, dim, k, knn_dist_gpu, knn_index_gpu, index_gpu, dist_gpu);
    
    // ---------------------------------- Transfering data on host -------------------------------

    gpuErrchk(hipMemcpy(knn_dist, knn_dist_gpu, query_nb*k*sizeof(float), hipMemcpyDeviceToHost));
    gpuErrchk(hipMemcpy(knn_index, knn_index_gpu, query_nb*k*sizeof(int), hipMemcpyDeviceToHost));
    

    // ---------------------------------- Debug section -------------------------------

    
    if(LOG_LEVEL < 2){   
        
        
        std::cout << "------------------------- Fill func -----------------------------";
        for(int query_index = 0; query_index < query_nb ; ++query_index){
            std::cout<< std::endl << query_index <<" query:" << std::endl; 

            for (int ref_index = 0; ref_index < ref_nb; ++ref_index){
                std::cout<< std::endl << "\t" <<ref_index <<" ref:" << std::endl;

                for(int d = 0; d < dim; d++){
                    
                    int it = d + ref_index * dim + query_index * ref_nb * dim;;
                
                    std::cout<< "\t\t" << d <<" dim -> it: " << it << " dots: " << dots[it] << std::endl;
                }
            }
        }
        

        std::cout << "\n------------------------- Sum dots -----------------------------";
        for(int query_index = 0; query_index < query_nb ; ++query_index){
            std::cout<< std::endl << query_index <<" query:" << std::endl; 
            for (int ref_index = 0; ref_index < ref_nb; ++ref_index){
                std::cout << "\treference index: " << ref_index << std::endl;
                std::cout << "\t\t\tsum_dots: " << sum_dots[query_index + ref_index*query_nb] <<std::endl;
                std::cout << "\t\tsum_denom_a: " << sum_denom_a[query_index + ref_index*query_nb] <<std::endl;
                std::cout << "\t\tsum_denom_b: " << sum_denom_b[query_index + ref_index*query_nb] <<std::endl;
                //float temp = sum_dots[query_index + ref_index*query_nb] / (sqrt(sum_denom_a[query_index + ref_index*query_nb]) * sqrt(sum_denom_b[query_index + ref_index*query_nb]));
                //std::cout << "\t\tcosine distance: " << temp << std::endl;
            }
        }
        /*
        std::cout << "------------------------- cosine distance-----------------------------";
        for(int i = 0; i < query_nb ; ++i){
            std::cout<< std::endl << i <<" query:" << std::endl; 
            for (int j = 0; j < ref_nb; ++j){
                std::cout << "\treference index: " << index_gpu[i + j*query_nb] << " dist: " << dist_gpu[i + j*query_nb] <<std::endl;
            }
        }
        */
        std::cout << "------------------------- finish YourSolution.c -----------------------------";
    }

    
    // ---------------------------------- Free memory -------------------------------

    hipFree(ref_gpu);
    hipFree(query_gpu);
    hipFree(knn_dist_gpu);
    hipFree(knn_index_gpu);

    hipFree(index_gpu);
    hipFree(dist_gpu);

    return true;
}